/**
 * This file contains the implementation of the Transformer architecture.
 */

#include "transformer.cuh"

/**
 * @brief Builds the Transformer architecture.
 * @param t The Transformer structure to be built.
 * @param checkpoint_path The path to the checkpoint file.
 * @param num_kv_heads The number of key-value heads, for GQA.
 */ 
void build_transformer(Transformer *t, char *checkpoint_path, int num_kv_heads) {
    // Open the checkpoint file
    FILE *file = fopen(checkpoint_path, "rb");
    if (!file) { fprintf(stderr, "Couldn't open file %s\n", checkpoint_path); exit(EXIT_FAILURE); }
    
    // Read the configuration from the file
    if (fread(&t->config, sizeof(Config), 1, file) != 1) { exit(EXIT_FAILURE); }
    fclose(file);

    // Set the number of key-value heads
    t->num_kv_heads = num_kv_heads;

    // Memory map the checkpoint file
    t->fd = open(checkpoint_path, O_RDONLY);
    if (t->fd == -1) { fprintf(stderr, "open failed\n"); exit(EXIT_FAILURE); }
    t->file_size = lseek(t->fd, 0, SEEK_END);
    t->data = (float*)mmap(NULL, t->file_size, PROT_READ, MAP_PRIVATE, t->fd, 0);
    if (t->data == MAP_FAILED) { fprintf(stderr, "mmap failed\n"); exit(EXIT_FAILURE); }
    
    // Point to the start of the weights in the mapped memory
    float* weights_ptr = t->data + sizeof(Config) / sizeof(float);

    // Map the weights to the appropriate structures
    memory_map_weights(&t->weights, &t->config, weights_ptr, 1);

    // Allocate memory for the run state
    malloc_run_state(&t->state, &t->config);
}

/**
 * @brief Frees the Transformer architecture.
 * @param t The Transformer structure to be freed.
 */
void free_transformer(Transformer *t) {
    // Unmap the memory-mapped file if it exists
    if (t->data != NULL) {
        munmap(t->data, t->file_size);
        close(t->fd);
    }
    // Free the run state
    free_run_state(&t->state);
}

/**
 * @brief Performs the forward pass of the Transformer architecture.
 * @param transformer The Transformer structure.
 * @param token The token to be processed.
 * @param pos The position of the token in the sequence.
 * @return The logits of the token.
 */ 
float* forward(Transformer *transformer, int token, int pos) {
    Config* p = &transformer->config;
    TransformerWeights* w = &transformer->weights;
    RunState* s = &transformer->state;
    float *x = s->x;
    int dim = p->dim;
    int kv_dim = (p->dim * transformer->num_kv_heads) / p->n_heads;
    int hidden_dim =  p->hidden_dim;
    int head_size = dim / p->n_heads;

    // Copy the token embedding to GPU memory
    float* content_row = w->token_embedding + token * dim;
    CUDA_CHECK(hipMemcpy(x, content_row, dim * sizeof(float), hipMemcpyHostToDevice));

    // Process each layer of the transformer
    for(int l = 0; l < p->n_layers; l++) {
        // Layer normalization before self-attention
        rmsnorm(s->xb, x, w->rms_att_weight + l*dim, dim);

        // Compute query, key, and value
        matmul(s->q, s->xb, w->wq + l*dim*dim, dim, dim);
        matmul(s->k, s->xb, w->wk + l*dim*kv_dim, dim, kv_dim);
        matmul(s->v, s->xb, w->wv + l*dim*kv_dim, dim, kv_dim);

        // Apply rotary positional embedding
        RoPe_rotation(pos, s, dim, kv_dim, head_size);

        // Compute attention
        int loff = l * p->max_seq_len * kv_dim;
        grouped_query_attention(pos, p, s, kv_dim, transformer->num_kv_heads, head_size, loff);

        // Compute output projection
        matmul(s->xb2, s->xb, w->wo + l*dim*dim, dim, dim);

        // Residual connection
        accum(x, s->xb2, dim);

        // Layer normalization before feed-forward network
        rmsnorm(s->xb, x, w->rms_ffn_weight + l*dim, dim);

        // Feed-forward network
        int ffn_dim = p->hidden_dim * 2; 
        swiglu(s, dim, ffn_dim); 

        // Output projection of feed-forward network
        matmul(s->xb, s->hb, w->w2 + l*dim*p->hidden_dim, p->hidden_dim, dim);

        // Residual connection
        accum(x, s->xb, dim);
    }

    // Final layer normalization
    rmsnorm(x, x, w->rms_final_weight, dim);

    // Compute logits
    matmul(s->logits_gpu, x, w->wcls, p->dim, p->vocab_size);
    CUDA_CHECK(hipMemcpy(s->logits, s->logits_gpu, p->vocab_size * sizeof(float), hipMemcpyDeviceToHost));
    return s->logits;
}

/**
 * @brief Allocates the RunState buffers.
 * @param s The RunState structure to be allocated.
 * @param p The Config structure.
 */ 
void malloc_run_state(RunState *s, Config *p) {
    int kv_dim = (p->dim * p->n_kv_heads) / p->n_heads;
    
    // Allocate memory for various tensors used in the forward pass
    CUDA_CHECK(hipMalloc((void **) &s->x, p->dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->xb, p->dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->xb2, p->dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->hb, p->hidden_dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->hb2, p->hidden_dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->q, p->dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->key_cache, p->n_layers * p->max_seq_len * kv_dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->value_cache, p->n_layers * p->max_seq_len * kv_dim * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->att, p->n_heads * p->max_seq_len * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **) &s->logits_gpu, p->vocab_size * sizeof(float)));
    s->logits = (float *) calloc(p->vocab_size, sizeof(float));

    // Check if all allocations were successful
    if (!s->x || !s->xb || !s->xb2 || !s->hb || !s->hb2 || !s->q
        || !s->key_cache || !s->value_cache || !s->att || !s->logits_gpu || !s->logits) {
        fprintf(stderr, "hipMalloc failed!\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * @brief Frees the RunState buffers.
 * @param s The RunState structure to be freed.
 */
void free_run_state(RunState *s) {
    // Free all allocated GPU memory
    CUDA_CHECK(hipFree(s->x));
    CUDA_CHECK(hipFree(s->xb));
    CUDA_CHECK(hipFree(s->xb2));
    CUDA_CHECK(hipFree(s->hb));
    CUDA_CHECK(hipFree(s->hb2));
    CUDA_CHECK(hipFree(s->q));
    CUDA_CHECK(hipFree(s->att));
    CUDA_CHECK(hipFree(s->logits_gpu));
    CUDA_CHECK(hipFree(s->key_cache));
    CUDA_CHECK(hipFree(s->value_cache));
    
    // Free CPU memory
    free(s->logits);
}

/**
 * @brief Maps the weights of the Transformer architecture.
 * @param w The TransformerWeights structure to be mapped.
 * @param p The Config structure.
 * @param ptr The pointer to the weights.
 * @param shared_weights Whether the weights are shared.
 */
void memory_map_weights(TransformerWeights *w, Config *p, float *ptr, int shared_weights) {
    int head_size = p->dim / p->n_heads;
    unsigned long long n_layers = p->n_layers;
    
    // Map token embeddings
    w->token_embedding = ptr;
    ptr += p->vocab_size * p->dim;
    
    // Map attention weights
    w->rms_att_weight = ptr;
    ptr += n_layers * p->dim;
    w->wq = ptr;
    ptr += n_layers * p->dim * (p->n_heads * head_size);
    w->wk = ptr;
    ptr += n_layers * p->dim * (p->n_kv_heads * head_size);
    w->wv = ptr;
    ptr += n_layers * p->dim * (p->n_kv_heads * head_size);
    w->wo = ptr;
    ptr += n_layers * (p->n_heads * head_size) * p->dim;
    
    // Map feed-forward network weights
    w->rms_ffn_weight = ptr;
    ptr += n_layers * p->dim;
    w->w1 = ptr;
    ptr += n_layers * p->dim * p->hidden_dim;
    w->w2 = ptr;
    ptr += n_layers * p->dim * p->hidden_dim;
    w->b1 = ptr;
    ptr += n_layers * p->hidden_dim;
    w->b2 = ptr;
    ptr += n_layers * p->hidden_dim;
    
    // Map final layer normalization weights
    w->rms_final_weight = ptr;
    ptr += p->dim;
    
    // Skip rope frequencies (if present)
    ptr += p->max_seq_len * head_size / 2; 
    ptr += p->max_seq_len * head_size / 2;
    
    // Map classifier weights (may be shared with token embeddings)
    w->wcls = shared_weights ? w->token_embedding : ptr;
}
