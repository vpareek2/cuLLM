#include "hip/hip_runtime.h"
#include "nn.cuh"

const int num_threads_large = 1024;

hipblasHandle_t g_cublas_handle = nullptr;

__device__ int divUp(int a, int b) {
    return (a - 1) / b + 1;
}

__global__ void rmsnorm_kernel(float *o, float *x, float *weight, int size, int elementsPerThread) {
    // parallel reduction of sum of squares via CUB
    float ss = 0.0f;
    for (int i = 0; i < elementsPerThread; i++) {
        int j = threadIdx.x + i * num_threads_large;
        if (j < size)
            ss += x[j] * x[j];
    }
    using BlockReduce = hipcub::BlockReduce<float, num_threads_large>;
    __shared__ typename BlockReduce::TempStorage temp;
    ss = BlockReduce(temp).Sum(ss);

    // serialization point to calculate normalization factor
    __shared__ float shared_ss;
    if (threadIdx.x == 0) {
        ss /= size;
        ss += 1e-5f;
        ss = 1.0f / sqrtf(ss);
        shared_ss = ss;
    }
    __syncthreads();
    ss = shared_ss;

    // normalize and scale
    for (int i = 0; i < elementsPerThread; i++) {
        int j = threadIdx.x + i * num_threads_large;
        if (j < size) {
            o[j] = weight[j] * (ss * x[j]);
        }
    }
}

void rmsnorm(float *o, float *x, float *weight, int size) {
    int elementsPerThread = divUp(size, num_threads_large);
    rmsnorm_kernel<<<1, num_threads_large>>>(o, x, weight, size, elementsPerThread);
}

__device__ void softmax_gpu(float *__restrict__ x, int size) {
    int tid = threadIdx.x;
    int step = blockDim.x;

    float max_val = tid < size ? x[tid] : 0;
    for (int i = tid + step; i < size; i += step) {
        if (x[i] > max_val) {
            max_val = x[i];
        }
    }
    using BlockReduce = hipcub::BlockReduce<float, num_threads_large>;
    __shared__ typename BlockReduce::TempStorage temp;
    __shared__ float shared_val;
    max_val = BlockReduce(temp).Reduce(max_val, hipcub::Max());
    if (threadIdx.x == 0) {
        shared_val = max_val;
    }
    __syncthreads();
    max_val = shared_val;

    float sum = 0.0f;
    for (int i = tid; i < size; i += step) {
        x[i] = expf(x[i] - max_val);
        sum += x[i];
    }
    sum = BlockReduce(temp).Sum(sum);
    if (threadIdx.x == 0) {
        shared_val = sum;
    }
    __syncthreads();
    sum = shared_val;

    for (int i = tid; i < size; i += step) {
        x[i] /= sum;
    }
}

void matmul(float *xout, float *x, float *w, int n, int d) {
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemv(g_cublas_handle, HIPBLAS_OP_T, n, d, &alpha, w, n, x, 1, &beta, xout, 1);
}

__global__ void RoPE_rotation_kernel(int pos, float *sq, float *sk, int kv_dim, int head_size) {
    int i = threadIdx.x * 2;
    int head_dim = i % head_size;
    float freq = 1.0f / powf(500000.0f, head_dim / (float) head_size); // 500,000 RoPE frequency hyperparameter as per llama3.1 paper
    float val = pos * freq;
    float fcr = cosf(val);
    float fci = sinf(val);
    int rotn = i < kv_dim ? 2 : 1;
    for (int v = 0; v < rotn; v++) {
        float *vec = v == 0 ? sq : sk;
        float v0 = vec[i];
        float v1 = vec[i + 1];
        vec[i] = v0 * fcr - v1 * fci;
        vec[i + 1] = v0 * fci + v1 * fcr;
    }
}

void RoPE_rotation(int pos, RunState *s, int dim, int kv_dim, int head_size) {
    RoPE_rotation_kernel<<<1, dim / 2>>>(pos, s->q, s->k, kv_dim, head_size);
}

__global__ void f_silu_elementwise_mul_w3_kernel(float *shb, float *shb2, int hidden_dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < hidden_dim) {
        float val = shb[i];
        // silu(x)=x*σ(x), where σ(x) is the logistic sigmoid
        val *= (1.0f / (1.0f + expf(-val)));
        // elementwise multiply with w3(x)
        val *= shb2[i];
        shb[i] = val;
    }
}

void f_silu_elementwise_mul_w3(RunState *s, int hidden_dim) {
    int num_threads_small = 256; // You might want to define this as a constant
    f_silu_elementwise_mul_w3_kernel<<<divUp(hidden_dim, num_threads_small), num_threads_small>>>(s->hb, s->hb2, hidden_dim);
}

__global__ void accum_kernel(float *a, float *b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        a[i] += b[i];
    }
}

void accum(float *a, float *b, int size) {
    int num_threads_small = 256; // You might want to define this as a constant
    accum_kernel<<<divUp(size, num_threads_small), num_threads_small>>>(a, b, size);
}

void create_cublas_handle() {
    hipblasStatus_t stat = hipblasCreate(&g_cublas_handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        exit(EXIT_FAILURE);
    }
}

void destroy_cublas_handle() {
    hipblasStatus_t stat = hipblasDestroy(g_cublas_handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS destruction failed\n");
        exit(EXIT_FAILURE);
    }
}
